#include "hip/hip_runtime.h"
// Copyright (C) 2019 Xiao Zhai
// 
// This file is part of CPP-Fluid-Particles.
// 
// CPP-Fluid-Particles is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
// 
// CPP-Fluid-Particles is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
// 
// You should have received a copy of the GNU General Public License
// along with CPP-Fluid-Particles.  If not, see <http://www.gnu.org/licenses/>.

#include <vector>
#include <memory>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_vector_types.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/fill.h>
#include <thrust/execution_policy.h>
#include "CUDAFunctions.cuh"
#include "DArray.h"
#include "Particles.h"
#include "SPHParticles.h"
#include "BaseSolver.h"
#include "SPHSystem.h"

SPHSystem::SPHSystem(
	std::shared_ptr<SPHParticles>& fluidParticles,
	std::shared_ptr<SPHParticles>& boundaryParticles,
	std::shared_ptr<BaseSolver>& solver,
	const float3 spaceSize,
	const float sphCellLength,
	const float sphSmoothingRadius,
	const float dt,
	const float sphM0,
	const float sphRho0,
	const float sphRhoBoundary,
	const float sphStiff,
	const float sphVisc,
	const float sphSurfaceTensionIntensity,
	const float sphAirPressure,
	const float3 sphG,
	const int3 cellSize)
	:_fluids(fluidParticles), _boundaries(boundaryParticles),
	_solver(solver),
	_spaceSize(spaceSize),
	_sphCellLength(sphCellLength),
	_sphSmoothingRadius(sphSmoothingRadius),
	_dt(dt),
	_sphRho0(sphRho0),
	_sphRhoBoundary(sphRhoBoundary),
	_sphStiff(sphStiff),
	_sphVisc(sphVisc),
	_sphSurfaceTensionIntensity(sphSurfaceTensionIntensity),
	_sphAirPressure(sphAirPressure),
	_sphG(sphG),
	_cellSize(cellSize),
	cellStartFluid(cellSize.x* cellSize.y* cellSize.z + 1),
	cellStartBoundary(cellSize.x* cellSize.y* cellSize.z + 1),
	bufferInt(max(totalSize(), cellSize.x* cellSize.y* cellSize.z + 1))
{
	// step 1: init boundary particles
	neighborSearch(_boundaries, cellStartBoundary);
	// step 2: calculate boundary particles' mass
	computeBoundaryMass();
	// step 3: init fluid particles
	thrust::fill(thrust::device, _fluids->getMassPtr(), _fluids->getMassPtr() + _fluids->size(), sphM0);
	neighborSearch(_fluids, cellStartFluid);
	// step 4: fill all fluid particles' properties by calling step()
	step();
}

__device__ void contributeBoundaryKernel(float* sum_kernel, int i, int cellID, float3* pos, int* cellStart, int3 cellSize, float radius)
{
	int j, end;
	if (cellID == (cellSize.x * cellSize.y * cellSize.z)) return;
	j = cellStart[cellID];	end = cellStart[cellID + 1];
	while (j < end)
	{
		*sum_kernel += cubic_spline_kernel(length(pos[i] - pos[j]), radius);
		j++;
	}
	return;
}

__global__ void computeBoundaryMass_CUDA(float* mass, float3* pos, int num, int* cellStart, int3 cellSize, float cellLength, float rhoB, float radius)
{
	unsigned int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if (i >= num) return;
	int3 cellPos = make_int3(pos[i] / cellLength);
	int cellID;
#pragma unroll
	for (int m = 0; m < 27; m++)
	{
		cellID = particlePos2cellIdx(cellPos + make_int3(m / 9 - 1, (m % 9) / 3 - 1, m % 3 - 1), cellSize);
		contributeBoundaryKernel(&mass[i], i, cellID, pos, cellStart, cellSize, radius);
	}
	mass[i] = rhoB / fmaxf(EPSILON, mass[i]);
	return;
}

void SPHSystem::computeBoundaryMass() {
	computeBoundaryMass_CUDA <<<(_boundaries->size() - 1) / block_size + 1, block_size >>> (
		_boundaries->getMassPtr(), _boundaries->getPosPtr(), _boundaries->size(),
		cellStartBoundary.addr(), _cellSize, _sphCellLength, _sphRhoBoundary, _sphSmoothingRadius);
}

void SPHSystem::neighborSearch(const std::shared_ptr<SPHParticles> &particles, DArray<int> &cellStart)
{
	int num = particles->size();
	mapParticles2Cells_CUDA <<<(num - 1) / block_size + 1, block_size >>> (particles->getParticle2Cell(), particles->getPosPtr(), _sphCellLength, _cellSize, num);
	CUDA_CALL(hipMemcpy(bufferInt.addr(), particles->getParticle2Cell(), sizeof(int) * num, hipMemcpyDeviceToDevice));
	thrust::sort_by_key(thrust::device, bufferInt.addr(), bufferInt.addr() + num, particles->getPosPtr());
	CUDA_CALL(hipMemcpy(bufferInt.addr(), particles->getParticle2Cell(), sizeof(int) * num, hipMemcpyDeviceToDevice));
	thrust::sort_by_key(thrust::device, bufferInt.addr(), bufferInt.addr() + num, particles->getVelPtr());

	thrust::fill(thrust::device, cellStart.addr(), cellStart.addr() + _cellSize.x * _cellSize.y * _cellSize.z + 1, 0);
	countingInCell_CUDA <<<(num - 1) / block_size + 1, block_size >>> (cellStart.addr(), particles->getParticle2Cell(), num);
	thrust::exclusive_scan(thrust::device, cellStart.addr(), cellStart.addr() + _cellSize.x * _cellSize.y * _cellSize.z + 1, cellStart.addr());
	return;
}

float SPHSystem::step()
{
	hipEvent_t start, stop;
	CUDA_CALL(hipEventCreate(&start));
	CUDA_CALL(hipEventCreate(&stop));
	CUDA_CALL(hipEventRecord(start, 0));

	neighborSearch(_fluids, cellStartFluid);
	try {
		_solver->step(_fluids, _boundaries, cellStartFluid, cellStartBoundary,
			_spaceSize, _cellSize, _sphCellLength, _sphSmoothingRadius,
			_dt, _sphRho0, _sphRhoBoundary, _sphStiff, _sphVisc, _sphG,
			_sphSurfaceTensionIntensity, _sphAirPressure);
		hipDeviceSynchronize(); CHECK_KERNEL();
	}
	catch (const char* s) {
		std::cout << s << std::endl;
	}
	catch (...) {
		std::cout << "Unknown Exception at "<<__FILE__<<": line "<<__LINE__ << std::endl;
	}

	float milliseconds;
	CUDA_CALL(hipEventRecord(stop, 0));
	CUDA_CALL(hipEventSynchronize(stop));
	CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));
	CUDA_CALL(hipEventDestroy(start));
	CUDA_CALL(hipEventDestroy(stop));
	return milliseconds;
}
